// https://github.com/thrust/thrust/blob/8551c97870cd722486ba7834ae9d867f13e299ad/examples/sum_rows.cu

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <iostream>

// convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns

  __host__ __device__
  linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
  T operator()(T i)
  {
    return i / C;
  }
};

const int NUM_REPS = 10;

hipEvent_t startEvent, stopEvent;
float ms;

int test(thrust::device_vector<int>& array, int R, int C)
{
  // int R = 5;     // number of rows
  // int C = 8;     // number of columns


  // allocate storage for row sums and indices
  thrust::device_vector<int> row_sums(R);
  thrust::device_vector<int> row_indices(R);

  thrust::reduce_by_key
      (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C)),
       thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C)) + (R*C),
       array.begin(),
       row_indices.begin(),
       row_sums.begin(),
       thrust::equal_to<int>(),
       thrust::plus<int>());

  hipEventRecord(startEvent, 0);

  for (int i = 0; i < NUM_REPS; i++) {
      // compute row sums by summing values with equal row indices
      thrust::reduce_by_key
          (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C)),
           thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C)) + (R*C),
           array.begin(),
           row_indices.begin(),
           row_sums.begin(),
           thrust::equal_to<int>(),
           thrust::plus<int>());
  }

  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&ms, startEvent, stopEvent);

  printf("%15.0f", (ms / NUM_REPS) * 1e3 );

  return 0;
}

void dothatbench(int THEPOWER, int start) {

//        int start=0;
        int end=THEPOWER;

        int total_elems = 1 << THEPOWER;

        printf("Benchmarking Thrust %i.%i.%i TotalElems=%i\n",
               THRUST_MAJOR_VERSION, THRUST_MINOR_VERSION, THRUST_SUBMINOR_VERSION, total_elems);

        thrust::default_random_engine rng;
        thrust::uniform_int_distribution<int> dist(10, 99);

        // initialize data
        thrust::device_vector<int> array(total_elems);
        for (size_t i = 0; i < array.size(); i++)
            array[i] = dist(rng);

        printf("initialized array\n");

        for(int powy=start; powy<=end; powy++) {
            int powx = THEPOWER-powy;

            int num_segments = 1 << powy;
            int segment_size = 1 << powx;

            char buf[16];
            snprintf(buf, 16, "2^%i 2^%i", powy, powx);
            printf("%15s", buf);

            test(array, num_segments, segment_size);
        }
}

int main(int argc, char** argv) {

    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    dothatbench(20, 0);
    dothatbench(26, 0);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
}
