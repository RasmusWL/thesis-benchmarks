#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 *
 * Original code and text by Sean Baxter, NVIDIA Research See
 * http://nvlabs.github.io/moderngpu for repository and documentation.
 *
 ******************************************************************************/

#include "kernels/segreducecsr.cuh"

using namespace mgpu;

enum TestType {
        TestTypeNormal,
        TestTypeIndirect,
        TestTypePreprocess
};

// count = total number of elements
//
template<typename T>
void TestCsrReduce(int count, int segSize, int numIterations,
        TestType testType, bool supportEmpty, CudaContext& context) {

#ifdef _DEBUG
        numIterations = 1;
#endif

        std::vector<int> segCountsHost, csrHost;
        int total = 0;
        int numValidRows = 0;
        while(total < count) {
                numValidRows += 0 != segSize;
                csrHost.push_back(total ? (csrHost.back() + segCountsHost.back()) : 0);
                segCountsHost.push_back(segSize);
                total += segSize;
        }
        int numRows = (int)segCountsHost.size();

        // FIXME: This one is only used for the indirect test. Not sure what it
        // does.
        std::vector<int> sourcesHost(numRows);
        for(int i = 0; i < numRows; ++i)
                sourcesHost[i] = Rand(0, max(0, count - segSize));

        MGPU_MEM(int) csrDevice = context.Malloc(csrHost);
        MGPU_MEM(int) sourcesDevice = context.Malloc(sourcesHost);

        // Generate random ints as input.
        std::vector<T> dataHost(count);
        for(int i = 0; i < count; ++i)
                dataHost[i] = (T)Rand(1, 9);
        MGPU_MEM(T) dataDevice = context.Malloc(dataHost);

        MGPU_MEM(T) resultsDevice = context.Malloc<T>(numRows);

        std::auto_ptr<SegReducePreprocessData> preprocessData;
        SegReduceCsrPreprocess<T>(count, csrDevice->get(), numRows, supportEmpty,
                &preprocessData, context);

        context.Start();
        for(int it = 0; it < numIterations; ++it) {
                if(TestTypeNormal == testType)
                        SegReduceCsr(dataDevice->get(), csrDevice->get(), count, numRows,
                                supportEmpty, resultsDevice->get(), (T)0, mgpu::plus<T>(),
                                context);
                else if(TestTypeIndirect == testType)
                        IndirectReduceCsr(dataDevice->get(), csrDevice->get(),
                                sourcesDevice->get(), count, numRows, supportEmpty,
                                resultsDevice->get(), (T)0, mgpu::plus<T>(), context);
                else
                        SegReduceApply(*preprocessData, dataDevice->get(), (T)0,
                                mgpu::plus<T>(), resultsDevice->get(), context);
        }
        double elapsed = context.Split();

        printf("%12.3lf microseconds\n", elapsed * 1e6);

        std::vector<T> resultsHost;
        resultsDevice->ToHost(resultsHost);

        std::vector<T> resultsRef(numRows);
        for(int row = 0; row < numRows; ++row) {
                int begin = csrHost[row];
                int end = (row + 1 < numRows) ? csrHost[row + 1] : count;
                int count = end - begin;

                begin = (TestTypeIndirect == testType) ? sourcesHost[row] : begin;
                end = begin + count;

                T x = 0;
                for(int i = begin; i < end; ++i)
                        x = x + dataHost[i];

                resultsRef[row] = x;
        }

        for(int i = 0; i < numRows; ++i) {
                if(resultsRef[i] != resultsHost[i]) {
                        printf("REDUCTION ERROR ON SEGMENT %d\n", i);
                        exit(0);
                }
        }
}

const int RWL_NUM_ITERATIONS = 10;

template<typename T>
void BenchmarkSegReduce(TestType testType, bool supportEmpty,
                         CudaContext& context, int THEPOWER) {

        const char* typeString;
        if(TestTypeNormal == testType) typeString = "seg";
        else if(TestTypeIndirect == testType) typeString = "indirect";
        else typeString = "preprocess";

        int start=0;
        int end=THEPOWER;

        int total_elems = 1 << THEPOWER;

        printf("Benchmarking %s-reduce type %s. TotalElems=%i\n",
               typeString, TypeIdName<T>(), total_elems);

        for(int powy=start; powy<=end; powy++) {
            int powx = THEPOWER-powy;

//            int num_segments = 1 << powy;
            int segment_size = 1 << powx;

            char buf[16];
            snprintf(buf, 16, "2^%i 2^%i", powy, powx);

            printf("%15s", buf);

            TestCsrReduce<T>(total_elems, segment_size, RWL_NUM_ITERATIONS, testType,
                             supportEmpty, context);

            context.GetAllocator()->Clear();
        }
        printf("\n");
}

int main(int argc, char** argv) {
        ContextPtr context = CreateCudaDevice(argc, argv, true);

        bool supportEmpty = false;
        TestType testType = TestTypeNormal;

        BenchmarkSegReduce<float>(testType, supportEmpty, *context, 20);
        BenchmarkSegReduce<float>(testType, supportEmpty, *context, 26);

        return 0;
}
